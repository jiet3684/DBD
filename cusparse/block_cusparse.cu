#include "cusparse.hpp"
#include <hipsparse.h>

void read_CSR(std::string , csr *);
void matMul(csr, csr, int);

int main (int argc, char **argv) {
    hipFree(0);

    csr A;
    csr B;

    read_CSR(argv[1], &A);
    read_CSR(argv[1], &B);

    //matMul(A, B, atoi(argv[1]));
    matMul(A, B, 1);

}

void read_CSR(std::string input_File, csr *mat) {
    if (access((input_File + ".csr").c_str(), F_OK) < 0) {
        printf("No Such File\n");
        exit(0);
    }

    FILE *fp;
    fp = fopen((input_File + ".csr").c_str(), "rb");
    int row, col, edge;
    fread(&row, sizeof(int), 1, fp);
    fread(&col, sizeof(int), 1, fp);
    fread(&edge, sizeof(int), 1, fp);
    mat->nr = row;
    mat->nc = col;
    mat->ne = edge;

    mat->ptr = (int*)malloc(sizeof(int) * (mat->nr + 1));
    mat->idx = (int*)malloc(sizeof(int) * mat->ne);
    mat->val = (float*)malloc(sizeof(float) * mat->ne);

    fread(mat->ptr, sizeof(int), mat->nr + 1, fp);
    fread(mat->idx, sizeof(int), mat->ne, fp);
    fread(mat->val, sizeof(float), mat->ne, fp);
    fclose(fp);

    //printf("Input: %s\n\tNumber of Rows:\t\t%d\n\tNumber of Columns:\t%d\n\tNumber of Edges:\t%d\n\n", input_File.c_str(), mat->nr, mat->nc, mat->ne);
}

void filewrite(int *ptr, int *idx, float *val, int nr, int64_t ne) {
    FILE *fp = fopen("output", "wb");

    fwrite(ptr, sizeof(int), nr + 1, fp);
    fwrite(idx, sizeof(int), ne, fp);
    fwrite(val, sizeof(float), ne, fp);

    fflush(fp);
    fclose(fp);
}


void matMul(csr A, csr B, int mode) {
    float t_kernel = 0, t_d2h = 0, t_write = 0;
    struct timeval st, ed;
    long nnzC = 0;

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseSpMatDescr_t descrA = 0;
    hipsparseSpMatDescr_t descrB = 0;
    hipsparseSpMatDescr_t descrC = 0;

    hipsparseSpGEMMDescr_t descr;

    hipsparseOperation_t op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    float cons = 1.0;

    hipsparseIndexType_t type1 = HIPSPARSE_INDEX_32I;
    hipsparseIndexType_t type2 = HIPSPARSE_INDEX_32I;
    hipsparseIndexBase_t base = HIPSPARSE_INDEX_BASE_ZERO;

    hipDataType dType = HIP_R_32F;

    int *ptrA, *idxA, *ptrB, *idxB;
    float *valA, *valB;

    hipMalloc((void**)&ptrA, sizeof(int) * (A.nr +1));
    hipMalloc((void**)&idxA, sizeof(int) * A.ne);
    hipMalloc((void**)&valA, sizeof(float) * A.ne);
    hipMalloc((void**)&ptrB, sizeof(int) * (B.nr +1));
    hipMalloc((void**)&idxB, sizeof(int) * B.ne);
    hipMalloc((void**)&valB, sizeof(float) * B.ne);

    hipMemcpy(ptrB, B.ptr, sizeof(int) * (B.nr + 1), hipMemcpyHostToDevice);
    hipMemcpy(idxA, A.idx, sizeof(int) * A.ne, hipMemcpyHostToDevice);
    hipMemcpy(valA, A.val, sizeof(float) * A.ne, hipMemcpyHostToDevice);
    hipMemcpy(idxB, B.idx, sizeof(int) * B.ne, hipMemcpyHostToDevice);
    hipMemcpy(valB, B.val, sizeof(float) * B.ne, hipMemcpyHostToDevice);

    int *h_ptrC, *h_idxC;
    float *h_valC;
    
    int *ptrC, *idxC;
    float *valC;
    
    size_t bufSize1;
    size_t bufSize2;
    char *buf1, *buf2;
    int *temp_ptrA = (int*)malloc(sizeof(int) * (BLOCKSIZE + 1));

    printf("cuSPARSE\n");
    for (int i = 0; i < ITER; ++i) {
        nnzC = 0;
        // Single Block
    
        float t1 = 0, t2 = 0, t3 = 0;
        int num_Blocks  = ((A.nr - 1) / BLOCKSIZE) + 1;


        for (int i = 0; i < num_Blocks; ++i) {
            hipsparseSpGEMM_createDescr(&descr);
            int nrA = BLOCKSIZE;
            int start_Row = i * BLOCKSIZE;
            int end_Row = (i + 1) * BLOCKSIZE;
            if (i == num_Blocks - 1) {
                nrA = A.nr - ((num_Blocks - 1) * BLOCKSIZE);
                end_Row = A.nr;
            }
            int *temp_idxA = idxA + A.ptr[i * BLOCKSIZE];
            float *temp_valA = valA + A.ptr[i * BLOCKSIZE];
            
            for (int j = 0; j < nrA + 1; ++j) {
                temp_ptrA[j] = A.ptr[start_Row + j] - A.ptr[start_Row];
            }
            hipMemcpy(ptrA, temp_ptrA, sizeof(int) * (nrA + 1), hipMemcpyHostToDevice);

            hipsparseCreateCsr(&descrA, nrA, A.nc, temp_ptrA[nrA], ptrA, idxA + A.ptr[start_Row], valA + A.ptr[start_Row], type1, type2, base, dType);
            hipsparseCreateCsr(&descrB, B.nr, B.nc, B.ne, ptrB, idxB, valB, type1, type2, base, dType);
            hipsparseCreateCsr(&descrC, nrA, B.nc, 0, NULL, NULL, NULL, type1, type2, base, dType);
            
            gettimeofday(&st, NULL);

            hipsparseSpGEMM_workEstimation(handle, op, op, &cons, descrA, descrB, &cons, descrC, dType, \
            HIPSPARSE_SPGEMM_DEFAULT, descr, &bufSize1, NULL);
            hipMalloc((void**)&buf1, sizeof(char) * bufSize1);

            hipsparseSpGEMM_workEstimation(handle, op, op, &cons, descrA, descrB, &cons, descrC, dType, \
                HIPSPARSE_SPGEMM_DEFAULT, descr, &bufSize1, buf1);

            hipsparseSpGEMM_compute(handle, op, op, &cons, descrA, descrB, &cons, descrC, dType, \
            HIPSPARSE_SPGEMM_DEFAULT, descr, &bufSize2, NULL);
            hipMalloc((void**)&buf2, sizeof(char) * bufSize2);

            hipsparseSpGEMM_compute(handle, op, op, &cons, descrA, descrB, &cons, descrC, dType, \
                HIPSPARSE_SPGEMM_DEFAULT, descr, &bufSize2, buf2);
            gettimeofday(&ed, NULL);
            t1 += (float)(ed.tv_sec - st.tv_sec) + 0.000001 * (float)(ed.tv_usec - st.tv_usec);

            int64_t row, col, edge;
            hipsparseSpMatGetSize(descrC, &row, &col, &edge);
            nnzC += edge;
            //printf("%d %d %d\n", row, col, edge);

            hipMalloc((void**)&ptrC, sizeof(int) * (row + 1));
            hipMalloc((void**)&idxC, sizeof(int) * edge);
            hipMalloc((void**)&valC, sizeof(float) * edge);

            hipsparseCsrSetPointers(descrC, ptrC, idxC, valC);

            hipsparseSpGEMM_copy(handle, op, op, &cons, descrA, descrB, &cons, descrC, dType, \
            HIPSPARSE_SPGEMM_DEFAULT, descr);
            hipDeviceSynchronize();

            hipFree(buf1);
            hipFree(buf2);
            hipsparseDestroySpMat(descrA);
            hipsparseDestroySpMat(descrB);
            hipsparseDestroySpMat(descrC);

            h_ptrC = (int*)malloc(sizeof(int) * (row + 1));
            h_idxC = (int*)malloc(sizeof(int) * edge);
            h_valC = (float*)malloc(sizeof(float) * edge);

            hipMemcpy(h_ptrC, ptrC, sizeof(int) * (row + 1), hipMemcpyDeviceToHost);
            hipMemcpy(h_idxC, idxC, sizeof(int) * edge, hipMemcpyDeviceToHost);
            hipMemcpy(h_valC, valC, sizeof(float) * edge, hipMemcpyDeviceToHost);

            hipDeviceSynchronize();
            gettimeofday(&ed, NULL);
            t2 += (float)(ed.tv_sec - st.tv_sec) + 0.000001 * (float)(ed.tv_usec - st.tv_usec);
            
            filewrite(h_ptrC, h_idxC, h_valC, row, edge);
            gettimeofday(&ed, NULL);
            t3 += (float)(ed.tv_sec - st.tv_sec) + 0.000001 * (float)(ed.tv_usec - st.tv_usec);
            
            free(h_ptrC);
            free(h_idxC);
            free(h_valC);
            

            hipFree(ptrC);
            hipFree(idxC);
            hipFree(valC);

            hipsparseSpGEMM_destroyDescr(descr);
        }
        printf("%f\t%f\t%f\n", t1, t2, t3);
        t_kernel += t1;
        t_d2h += t2;
        t_write += t3;
        
    }
    printf("\nTotal nnzC %ld\n", nnzC);
    printf("Average:\n%f\t%f\t%f\n", t_kernel / ITER, t_d2h / ITER, t_write / ITER);
    hipsparseDestroy(handle);

        free(temp_ptrA);
    free(A.ptr);
    free(A.idx);
    free(A.val);
    free(B.ptr);
    free(B.idx);
    free(B.val);

    hipFree(ptrA);
    hipFree(idxA);
    hipFree(valA);
    hipFree(ptrB);
    hipFree(idxB);
    hipFree(valB);


}
