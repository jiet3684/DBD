#include "cusparse.hpp"
#include <hipsparse.h>

void read_CSR(std::string , csr *);
void matMul(csr, csr, int);

int main (int argc, char **argv) {
    hipFree(0);

    csr A;
    csr B;

    read_CSR(argv[1], &A);
    read_CSR(argv[1], &B);

    //matMul(A, B, atoi(argv[1]));
    matMul(A, B, 0);

}

void read_CSR(std::string input_File, csr *mat) {
    if (access((input_File + ".csr").c_str(), F_OK) < 0) {
        printf("No Such File\n");
        exit(0);
    }

    FILE *fp;
    fp = fopen((input_File + ".csr").c_str(), "rb");
    int row, col, edge;
    fread(&row, sizeof(int), 1, fp);
    fread(&col, sizeof(int), 1, fp);
    fread(&edge, sizeof(int), 1, fp);
    mat->nr = row;
    mat->nc = col;
    mat->ne = edge;

    mat->ptr = (int*)malloc(sizeof(int) * (mat->nr + 1));
    mat->idx = (int*)malloc(sizeof(int) * mat->ne);
    mat->val = (float*)malloc(sizeof(float) * mat->ne);

    fread(mat->ptr, sizeof(int), mat->nr + 1, fp);
    fread(mat->idx, sizeof(int), mat->ne, fp);
    fread(mat->val, sizeof(float), mat->ne, fp);
    fclose(fp);

    //printf("Input: %s\n\tNumber of Rows:\t\t%d\n\tNumber of Columns:\t%d\n\tNumber of Edges:\t%d\n\n", input_File.c_str(), mat->nr, mat->nc, mat->ne);
}

void filewrite(int *ptr, int *idx, float *val, int nr, int64_t ne) {
    FILE *fp = fopen("output", "wb");

    fwrite(ptr, sizeof(int), nr + 1, fp);
    fwrite(idx, sizeof(int), ne, fp);
    fwrite(val, sizeof(float), ne, fp);

    fclose(fp);
}


void matMul(csr A, csr B, int mode) {
    float t_kernel = 0, t_d2h = 0, t_write = 0;
    struct timeval st, ed;
    long nnzC = 0;

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseSpMatDescr_t descrA = 0;
    hipsparseSpMatDescr_t descrB = 0;
    hipsparseSpMatDescr_t descrC = 0;

    hipsparseSpGEMMDescr_t descr;

    hipsparseOperation_t op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    float cons = 1.0;

    hipsparseIndexType_t type1 = HIPSPARSE_INDEX_32I;
    hipsparseIndexType_t type2 = HIPSPARSE_INDEX_32I;
    hipsparseIndexBase_t base = HIPSPARSE_INDEX_BASE_ZERO;

    hipDataType dType = HIP_R_32F;

    int *ptrA, *idxA, *ptrB, *idxB;
    float *valA, *valB;

    hipMalloc((void**)&ptrA, sizeof(int) * (A.nr +1));
    hipMalloc((void**)&idxA, sizeof(int) * A.ne);
    hipMalloc((void**)&valA, sizeof(float) * A.ne);
    hipMalloc((void**)&ptrB, sizeof(int) * (B.nr +1));
    hipMalloc((void**)&idxB, sizeof(int) * B.ne);
    hipMalloc((void**)&valB, sizeof(float) * B.ne);

    hipMemcpy(ptrB, B.ptr, sizeof(int) * (B.nr + 1), hipMemcpyHostToDevice);
    hipMemcpy(idxA, A.idx, sizeof(int) * A.ne, hipMemcpyHostToDevice);
    hipMemcpy(valA, A.val, sizeof(float) * A.ne, hipMemcpyHostToDevice);
    hipMemcpy(idxB, B.idx, sizeof(int) * B.ne, hipMemcpyHostToDevice);
    hipMemcpy(valB, B.val, sizeof(float) * B.ne, hipMemcpyHostToDevice);

    int *h_ptrC, *h_idxC;
    float *h_valC;
    
    int *ptrC, *idxC;
    float *valC;
    
    size_t bufSize1;
    size_t bufSize2;
    char *buf1, *buf2;

    printf("cuSPARSE\n");
    for (int i = 0; i < ITER + 1; ++i) {
        nnzC = 0;
        // Single Block
        hipsparseSpGEMM_createDescr(&descr);

        hipMemcpy(ptrA, A.ptr, sizeof(int) * (A.nr + 1), hipMemcpyHostToDevice);

        hipsparseCreateCsr(&descrA, A.nr, A.nc, A.ne, ptrA, idxA, valA, type1, type2, base, dType);
        hipsparseCreateCsr(&descrB, B.nr, B.nc, B.ne, ptrB, idxB, valB, type1, type2, base, dType);
        hipsparseCreateCsr(&descrC, A.nr, B.nc, 0, NULL, NULL, NULL, type1, type2, base, dType);
        

        gettimeofday(&st, NULL);
        hipsparseSpGEMM_workEstimation(handle, op, op, &cons, descrA, descrB, &cons, descrC, dType, \
        HIPSPARSE_SPGEMM_DEFAULT, descr, &bufSize1, NULL);
        hipMalloc((void**)&buf1, sizeof(char) * bufSize1);

        hipsparseSpGEMM_workEstimation(handle, op, op, &cons, descrA, descrB, &cons, descrC, dType, \
            HIPSPARSE_SPGEMM_DEFAULT, descr, &bufSize1, buf1);

        hipsparseSpGEMM_compute(handle, op, op, &cons, descrA, descrB, &cons, descrC, dType, \
        HIPSPARSE_SPGEMM_DEFAULT, descr, &bufSize2, NULL);
        hipMalloc((void**)&buf2, sizeof(char) * bufSize2);

        hipsparseSpGEMM_compute(handle, op, op, &cons, descrA, descrB, &cons, descrC, dType, \
            HIPSPARSE_SPGEMM_DEFAULT, descr, &bufSize2, buf2);

        gettimeofday(&ed, NULL);
        if (i > 0) if (i > 0) t_kernel += (float)(ed.tv_sec - st.tv_sec) + 0.000001 * (float)(ed.tv_usec - st.tv_usec);
        printf("%f\t", (float)(ed.tv_sec - st.tv_sec) + 0.000001 * (float)(ed.tv_usec - st.tv_usec));
        int64_t row, col, edge;
        hipsparseSpMatGetSize(descrC, &row, &col, &edge);
        nnzC = edge;

        hipMalloc((void**)&ptrC, sizeof(int) * (row + 1));
        hipMalloc((void**)&idxC, sizeof(int) * edge);
        hipMalloc((void**)&valC, sizeof(float) * edge);

        hipsparseCsrSetPointers(descrC, ptrC, idxC, valC);

        hipsparseSpGEMM_copy(handle, op, op, &cons, descrA, descrB, &cons, descrC, dType, \
        HIPSPARSE_SPGEMM_DEFAULT, descr);
        hipDeviceSynchronize();
        hipsparseSpGEMM_destroyDescr(descr);

        h_ptrC = (int*)malloc(sizeof(int) * (row + 1));
        h_idxC = (int*)malloc(sizeof(int) * edge);
        h_valC = (float*)malloc(sizeof(float) * edge);

        hipMemcpy(h_ptrC, ptrC, sizeof(int) * (row + 1), hipMemcpyDeviceToHost);
        hipMemcpy(h_idxC, idxC, sizeof(int) * edge, hipMemcpyDeviceToHost);
        hipMemcpy(h_valC, valC, sizeof(float) * edge, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();
        gettimeofday(&ed, NULL);
        if (i > 0) t_d2h += (float)(ed.tv_sec - st.tv_sec) + 0.000001 * (float)(ed.tv_usec - st.tv_usec);
        printf("%f\t", (float)(ed.tv_sec - st.tv_sec) + 0.000001 * (float)(ed.tv_usec - st.tv_usec));
        
        filewrite(h_ptrC, h_idxC, h_valC, A.nr, edge);
        gettimeofday(&ed, NULL);
        if (i > 0) t_write += (float)(ed.tv_sec - st.tv_sec) + 0.000001 * (float)(ed.tv_usec - st.tv_usec);
        printf("%f\n", (float)(ed.tv_sec - st.tv_sec) + 0.000001 * (float)(ed.tv_usec - st.tv_usec));
        
        free(h_ptrC);
        free(h_idxC);
        free(h_valC);
        
        hipFree(buf1);
        hipFree(buf2);
        hipFree(ptrC);
        hipFree(idxC);
        hipFree(valC);
    
        
    }
    printf("\nTotal nnzC %ld\n", nnzC);
    printf("Average:\n%f\t%f\t%f\n", t_kernel / ITER, t_d2h / ITER, t_write / ITER);
    hipsparseDestroy(handle);

    free(A.ptr);
    free(A.idx);
    free(A.val);
    free(B.ptr);
    free(B.idx);
    free(B.val);

    hipFree(ptrA);
    hipFree(idxA);
    hipFree(valA);
    hipFree(ptrB);
    hipFree(idxB);
    hipFree(valB);


}
